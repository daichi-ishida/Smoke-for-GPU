#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <math.h>

#include "constants.h"
#include "Smoke.h"

#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>

#include <algorithm>

#include "hip/hip_runtime_api.h"

#define FOR_EACH_CELL  \
    for (int k = 0; k < zRes; ++k) \
        for (int j = 0; j < yRes; ++j) \
            for (int i = 0; i < xRes; ++i)

#define FOR_EACH_FACE_X  \
    for (int k = 0; k < zRes; ++k) \
        for (int j = 0; j < yRes; ++j) \
            for (int i = 0; i < xRes + 1; ++i)

#define FOR_EACH_FACE_Y  \
    for (int k = 0; k < zRes; ++k) \
        for (int j = 0; j < yRes + 1; ++j) \
            for (int i = 0; i < xRes; ++i)

#define FOR_EACH_FACE_Z  \
    for (int k = 0; k < zRes + 1; ++k) \
        for (int j = 0; j < yRes; ++j) \
            for (int i = 0; i < xRes; ++i)

Smoke::Smoke()
    : dt(0.0f), t(0.0f), next_shutter_time(0.0f), isTimeToRender(false)
{
    //dt = std::min(CFL * DX / INIT_VELOCITY, 1.0f / FPS);
    dt = DT;

    if (SAVE_VDB)
    {
        openvdb::initialize();
    }
}

Smoke::~Smoke()
{
}

void Smoke::initialize()
{
    // initial setup
    printf("initializing data...");
    initVelocity();
    initDensity();
    initTemperature();
    setObstacles();

    d_force_y_data.resize(xRes * yRes * zRes);
    d_pressure_data.resize(xRes * yRes * zRes);
    d_Ax_data.resize(xRes * yRes * zRes);
    d_divergence_data.resize(xRes * yRes * zRes);
    d_direction_data.resize(xRes * yRes * zRes);

    force_y.data = d_force_y_data.data();
    pressure.data = d_pressure_data.data();
    Ax.data = d_Ax_data.data();
    divergence.data = d_divergence_data.data();
    direction.data = d_direction_data.data();

    d_speed.resize(blockArraySize);

    printf("Done\n");
}

void Smoke::setNextShutterTime()
{
    next_shutter_time = t + 1.0f / FPS;
}


void Smoke::addScalarFieldForVDB(openvdb::GridPtrVec& grids, std::string name, const ScalarField& field) const
{
    openvdb::FloatGrid::Ptr scalarGrid = openvdb::FloatGrid::create(0.0f);
    scalarGrid->setName(name);
    scalarGrid->setTransform(openvdb::math::Transform::createLinearTransform(DX));
    scalarGrid->setGridClass(openvdb::GRID_FOG_VOLUME);

    auto scalarAccessor = scalarGrid->getAccessor();

    thrust::host_vector<float> h_data(xRes * yRes * zRes);

    thrust::copy_n(field.data, xRes * yRes * zRes, h_data.data());

    FOR_EACH_CELL
    {
        int offset = i + (j + k * yRes) * xRes;
        openvdb::Coord xyz(i, yRes - 1 - j, k);
        float value = h_data[offset];
        scalarAccessor.setValue(xyz, value);
    }

    grids.push_back(scalarGrid);
}

void Smoke::addUFieldForVDB(openvdb::GridPtrVec& grids, const uField& field) const
{
    openvdb::FloatGrid::Ptr staggeredGrid = openvdb::FloatGrid::create(0.0);
    staggeredGrid->setName("vel.x");
    staggeredGrid->setTransform(openvdb::math::Transform::createLinearTransform(DX));
    staggeredGrid->setGridClass(openvdb::GRID_STAGGERED);

    auto velAccessor = staggeredGrid->getAccessor();

    thrust::host_vector<float> h_data((xRes + 1) * yRes * zRes);

    thrust::copy_n(field.data, (xRes + 1) * yRes * zRes, h_data.data());

    FOR_EACH_FACE_X
    {
        int offset = i + (j + k * yRes) * (xRes + 1);
        openvdb::Coord xyz(i, yRes - 1 - j, k);
        float value = h_data[offset];
        velAccessor.setValue(xyz, value);
    }

    grids.push_back(staggeredGrid);

}

void Smoke::addVFieldForVDB(openvdb::GridPtrVec& grids, const vField& field) const
{
    openvdb::FloatGrid::Ptr staggeredGrid = openvdb::FloatGrid::create(0.0);
    staggeredGrid->setName("vel.y");
    staggeredGrid->setTransform(openvdb::math::Transform::createLinearTransform(DX));
    staggeredGrid->setGridClass(openvdb::GRID_STAGGERED);

    auto velAccessor = staggeredGrid->getAccessor();

    thrust::host_vector<float> h_data(xRes * (yRes + 1) * zRes);

    thrust::copy_n(field.data, xRes * (yRes + 1) * zRes, h_data.data());

    FOR_EACH_FACE_Y
    {
        int offset = i + (j + k * (yRes+1)) * xRes;
        openvdb::Coord xyz(i, yRes - j, k);
        float value = h_data[offset];
        velAccessor.setValue(xyz, value);
    }

    grids.push_back(staggeredGrid);
}

void Smoke::addWFieldForVDB(openvdb::GridPtrVec& grids, const wField& field) const
{
    openvdb::FloatGrid::Ptr staggeredGrid = openvdb::FloatGrid::create(0.0);
    staggeredGrid->setName("vel.z");
    staggeredGrid->setTransform(openvdb::math::Transform::createLinearTransform(DX));
    staggeredGrid->setGridClass(openvdb::GRID_STAGGERED);

    auto velAccessor = staggeredGrid->getAccessor();

    thrust::host_vector<float> h_data(xRes * yRes * (zRes + 1));

    thrust::copy_n(field.data, xRes * yRes * (zRes + 1), h_data.data());

    FOR_EACH_FACE_Z
    {
        int offset = i + (j + k * yRes) * xRes;
        openvdb::Coord xyz(i, yRes - 1 - j, k);
        float value = h_data[offset];
        velAccessor.setValue(xyz, value);
    }

    grids.push_back(staggeredGrid);
}

void Smoke::saveVDB() const
{
    static int count = 0;
    std::string filename;
    std::stringstream ss;
    ss << "VDB/res" << xRes << "x" << yRes << "x" << zRes << "_" << std::setfill('0') << std::setw(4) << std::right << std::to_string(count++) << ".vdb";
    ss >> filename;

    openvdb::GridPtrVec grids;

    addScalarFieldForVDB(grids, "density", density0);
    addScalarFieldForVDB(grids, "temperature", temperature0);
    addScalarFieldForVDB(grids, "pressure", pressure);
    addUFieldForVDB(grids, u0);
    addVFieldForVDB(grids, v0);
    addWFieldForVDB(grids, w0);

    // Create a VDB file object.
    openvdb::io::File file(filename);

    // Write out the contents of the container.
    file.write(grids);
    file.close();
}


void Smoke::initVelocity()
{
    thrust::host_vector<float> h_u0_scanline((xRes + 1) * yRes * zRes);
    thrust::host_vector<float> h_v0_scanline(xRes * (yRes + 1) * zRes);
    thrust::host_vector<float> h_w0_scanline(xRes * yRes * (zRes + 1));

    d_u0_data.resize((xRes + 1) * yRes * zRes);
    d_v0_data.resize(xRes * (yRes + 1) * zRes);
    d_w0_data.resize(xRes * yRes * (zRes + 1));

    d_u_data.resize((xRes + 1) * yRes * zRes);
    d_v_data.resize(xRes * (yRes + 1) * zRes);
    d_w_data.resize(xRes * yRes * (zRes + 1));

    // u
    OPENMP_FOR_COLLAPSE
    FOR_EACH_FACE_X
    {
        int offset = i + j * (xRes + 1) + k * (xRes + 1) * yRes;
        h_u0_scanline[offset] = 0.0f;
    }

    // v
    OPENMP_FOR_COLLAPSE
    FOR_EACH_FACE_Y
    {
        int offset = i + (j + k * (yRes + 1)) * xRes;
        h_v0_scanline[offset] = 0.0f;
        int dx = i - SOURCE_CENTER_X;
        int dz = k - SOURCE_CENTER_Z;
        int d2xz = dx * dx + dz * dz;
        if(j >= yRes - SOURCE_SIZE_Y && d2xz < SOURCE_RADIUS_XZ * SOURCE_RADIUS_XZ)
        {
            h_v0_scanline[offset] = INIT_VELOCITY;
        }
    }

        // w
    OPENMP_FOR_COLLAPSE
        FOR_EACH_FACE_Z
    {
        int offset = i + (j + k * yRes) * xRes;
        h_w0_scanline[offset] = 0.0f;
    }

    d_u0_data = h_u0_scanline;
    d_v0_data = h_v0_scanline;
    d_w0_data = h_w0_scanline;

    u0.data = d_u0_data.data();
    v0.data = d_v0_data.data();
    w0.data = d_w0_data.data();

    u.data = d_u_data.data();
    v.data = d_v_data.data();
    w.data = d_w_data.data();
}

void Smoke::initDensity()
{
    thrust::host_vector<float> h_density0_scanline(xRes * yRes * zRes);

    d_density0_data.resize(xRes * yRes * zRes);
    d_density_data.resize(xRes * yRes * zRes);

    OPENMP_FOR_COLLAPSE
        FOR_EACH_CELL
    {
        int offset = i + (j + k * yRes) * xRes;
        int dx = i - SOURCE_CENTER_X;
        int dz = k - SOURCE_CENTER_Z;
        int d2xz = dx * dx + dz * dz;
        if(j >= yRes - SOURCE_SIZE_Y && d2xz < SOURCE_RADIUS_XZ * SOURCE_RADIUS_XZ)
        {
            h_density0_scanline[offset] = INIT_DENSITY;
        }
    }

    d_density0_data = h_density0_scanline;

    density0.data = d_density0_data.data();
    density.data = d_density_data.data();
}

void Smoke::initTemperature()
{
    thrust::host_vector<float> h_temperature0_scanline(xRes * yRes * zRes);

    d_temperature0_data.resize(xRes * yRes * zRes);
    d_temperature_data.resize(xRes * yRes * zRes);

    OPENMP_FOR_COLLAPSE
        FOR_EACH_CELL
    {
        int offset = i + (j + k * yRes) * xRes;
        int dx = i - SOURCE_CENTER_X;
        int dz = k - SOURCE_CENTER_Z;
        int d2xz = dx * dx + dz * dz;
        h_temperature0_scanline[offset] = 0.0f;
        if(j >= yRes - SOURCE_SIZE_Y && d2xz < SOURCE_RADIUS_XZ * SOURCE_RADIUS_XZ)
        {
            h_temperature0_scanline[offset] = INIT_TEMPERATURE;
        }
    }

    d_temperature0_data = h_temperature0_scanline;

    temperature0.data = d_temperature0_data.data();
    temperature.data = d_temperature_data.data();
}

void Smoke::setObstacles()
{
    std::vector<char> h_scanline_obstacles(xRes * yRes * zRes);
    d_obstacles_data.resize(xRes * yRes * zRes);

    std::string filename = "resources/fan2_" + std::to_string(DIM) + ".sdf";
    std::ifstream fin( filename.c_str(), std::ios::in | std::ios::binary );
  
    fin.read(h_scanline_obstacles.data(), sizeof(char) * xRes * yRes * zRes);
    fin.close();

    d_obstacles_data = h_scanline_obstacles;

    obstacles.data = d_obstacles_data.data();
}



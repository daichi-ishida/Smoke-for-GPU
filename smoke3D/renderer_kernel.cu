#include "hip/hip_runtime.h"
#include "Smoke.h"
#include "renderer.h"
#include "constants.h"

#include "renderer.cuh"
#include "camera.h"

#include <cuda_gl_interop.h>

#include "hip/hip_runtime_api.h"
#include "hip/hip_vector_types.h"
#include "helper_cudaVS.h"


texture<float, 3, hipReadModeElementType> densityTex;

__constant__ Camera c_camera;

__global__ void render_k(uchar4* d_output, const Obstacles obstacles)
{
    const float tstep = 2.0f / (float)DIM;
    float3 boxMin = make_float3(-1.0f, -2.0f, -1.0f);
    float3 boxMax = make_float3(1.0f, 2.0f, 1.0f);
    float3 albedo = make_float3(0.4f, 0.4f, 0.4f);

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if ((x >= WIN_WIDTH) || (y >= WIN_HEIGHT)) return;

    Ray eyeRay = c_camera.generateRay(x, y);

    // find intersection with box to skip empty place
    float tnear, tfar;
    bool hit = isIntersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

    if (!hit) return;

    if (tnear < 0.0f) tnear = 0.0f; // clamp to near plane
    float t = tnear;
    float3 ray_wpos = eyeRay.o + eyeRay.d * tnear;
    float3 du = eyeRay.d * tstep;

    // float3 lightPos = make_float3(7.0f, -8.5f, 7.0f);
    float r = 7.0f;
    float verticalAngle = M_PI / 3.0f;
    float horizontalAngle = 0.0f;
    float3 lightPos = make_float3(r * sin(verticalAngle) * sin(horizontalAngle),
        -r * cos(verticalAngle),
        r * sin(verticalAngle) * cos(horizontalAngle));
    const float lightIntensity = 10.0f;

    // transmittance
    float3 T = make_float3(1.0f);
    float3 Lo = make_float3(0.0f);

    bool hitObstacle = false;
    bool hitWall = false;

    // while ray is in boundary box
    for (; ; t += tstep, ray_wpos += du)
    {
        if (t > tfar)
        {
            hitWall = true;
            break;
        }
        // world position to grid position
        float3 gridPos = convertToGridCoordinate(ray_wpos);

        // check intersection with obstacle
        if (obstacles.globalSmapler(gridPos.x, gridPos.y, gridPos.z))
        {
            hitObstacle = true;
            break;
        }

        // sample density
        float density = 200.0f * tex3D(densityTex, gridPos.x, gridPos.y, gridPos.z);

        // skip empty space
        if (density <= 0.0f) continue;
        float3 sigma_scatter = albedo * density;
        float3 sigma_absorption = make_float3(density) - sigma_scatter;
        float3 sigma_total = sigma_scatter + sigma_absorption;
        float3 omega = sigma_scatter / sigma_total;

        T *= expf(-sigma_total * tstep);
        if (T.x < 0.001f && T.y < 0.001f && T.z < 0.001f) break;

        float3 lightDir = normalize(lightPos - ray_wpos);
        float3 light_ray_wpos = ray_wpos;
        float3 dx = lightDir * tstep;

        // transmittance along light ray
        float3 lightT = make_float3(1.0f);

        for (; ; light_ray_wpos += dx)
        {
            // world position to grid position
            float3 light_gridPos = convertToGridCoordinate(light_ray_wpos);

            // check ray is in box
            if (!isInGridBox(light_gridPos)) break;

            // sample density
            float light_density = 200.0f * tex3D(densityTex, light_gridPos.x, light_gridPos.y, light_gridPos.z);

            // skip empty space
            if (light_density <= 0.0f) continue;
            float3 light_sigma_scatter = albedo * light_density;
            float3 light_sigma_absorption = make_float3(light_density) - light_sigma_scatter;
            float3 light_sigma_total = light_sigma_scatter + light_sigma_absorption;

            lightT *= expf(-light_sigma_total * tstep);
            if (lightT.x < 0.001f && lightT.y < 0.001f && lightT.z < 0.001f) break;
        }
        float3 Jss = lightIntensity * lightT * omega * mieScatter(lightDir, -eyeRay.d, 0.2f);
        Lo += T * sigma_total * Jss * tstep;
    }

    float alpha = 1.0f - grayScale(T);
    if (hitObstacle)
    {
        Lo += T * make_float3(0.2f, 0.2f, 0.3f);
        alpha = 1.0f;
    }
    else if (hitWall)
    {
        Lo += T * drawWall(ray_wpos, lightPos, -eyeRay.d);
        alpha = 1.0f;
    }

    Lo = fminf(Lo, make_float3(1.0f));

    d_output[offset].x = Lo.x * 255;
    d_output[offset].y = Lo.y * 255;
    d_output[offset].z = Lo.z * 255;
    d_output[offset].w = alpha * 255;
}

Renderer::~Renderer()
{
    if (pbo)
    {
        // unregister this buffer object from CUDA C
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        // delete old buffer
        glDeleteBuffers(1, &pbo);
        glDeleteTextures(1, &tex_buffer);
    }
    unbindDensityTexture();
}

void Renderer::initialize()
{
    glEnable(GL_TEXTURE_2D);

    if (pbo)
    {
        printf("unregister buffer...");
        // unregister this buffer object from CUDA C
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        // delete old buffer
        glDeleteBuffers(1, &pbo);
        glDeleteTextures(1, &tex_buffer);
        printf("Done\n");
    }

    printf("generating buffer...");

    // create pixel buffer object for display
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, WIN_WIDTH * WIN_HEIGHT * 4, 0, GL_DYNAMIC_DRAW_ARB);

    // register this buffer object with CUDA
    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo, cudaGraphicsMapFlagsWriteDiscard);

    // create texture for display
    glGenTextures(1, &tex_buffer);
    glBindTexture(GL_TEXTURE_2D, tex_buffer);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, WIN_WIDTH, WIN_HEIGHT, 0, GL_RGBA, GL_UNSIGNED_BYTE, NULL);

    glBindTexture(GL_TEXTURE_2D, 0);
    getLastCudaError("Renderer initializing failed\n");
    printf("Done\n");

    printf("setting camera...");
    host_camera = std::make_unique<Camera>(9.0f, M_PIf / 9.0f, M_PIf / 2.0f, 30.0f * M_PIf / 180.0f);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(c_camera), host_camera.get(), sizeof(Camera)));
    printf("Done\n");

    bindDensityTexture();
}

void Renderer::render()
{
    // map PBO to get CUDA device pointer
    uchar4* dev_ptr;
    // map PBO to get CUDA device pointer
    checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
    size_t num_bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dev_ptr, &num_bytes, cuda_pbo_resource));

    // clear image
    checkCudaErrors(hipMemset(dev_ptr, 0, WIN_WIDTH * WIN_HEIGHT * 4));

    // call CUDA kernel, writing results to PBO
    dim3 blocks((WIN_WIDTH + 31) / 32, (WIN_HEIGHT + 31) / 32);
    dim3 render_threads(32, 32);

    assignTexture();

    CALL_KERNEL(render_k, blocks, render_threads)(dev_ptr, m_data->obstacles);

    getLastCudaError("render failed\n");

    checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

    // display results
    glClear(GL_COLOR_BUFFER_BIT);

    // draw image from PBO
    glDisable(GL_DEPTH_TEST);

    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);

    // glDrawPixels is slow so use texture instead

    // copy from pbo to texture
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, pbo);
    glBindTexture(GL_TEXTURE_2D, tex_buffer);

    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, WIN_WIDTH, WIN_HEIGHT, GL_RGBA, GL_UNSIGNED_BYTE, 0);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    // draw textured quad
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(-1.0f, -1.0f);
    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(1.0f, -1.0f);
    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(-1.0f, 1.0f);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
}

void Renderer::bindDensityTexture()
{
    hipChannelFormatDesc cdesc = hipCreateChannelDesc<float>();
    checkCudaErrors(hipMalloc3DArray(&cuda_density, &cdesc, make_hipExtent(xRes, yRes, zRes)));
    getLastCudaError("hipMallocArray failed\n");

    densityTex.filterMode = hipFilterModeLinear;
    densityTex.addressMode[0] = hipAddressModeClamp;
    densityTex.addressMode[1] = hipAddressModeClamp;
    densityTex.addressMode[2] = hipAddressModeClamp;

    checkCudaErrors(hipBindTextureToArray(densityTex, cuda_density, cdesc));
    getLastCudaError("hipBindTextureToArray failed\n");
}

void Renderer::unbindDensityTexture()
{
    checkCudaErrors(hipUnbindTexture(densityTex));
}

void Renderer::assignTexture()
{
    hipMemcpy3DParms parms = { 0 };

    parms.dstArray = cuda_density;
    parms.srcPtr = make_hipPitchedPtr(thrust::raw_pointer_cast(m_data->density0.data), sizeof(float) * xRes, xRes, yRes);
    parms.extent = make_hipExtent(xRes, yRes, zRes);
    parms.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&parms));
}
